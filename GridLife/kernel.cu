#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <stdio.h>

#include <stdlib.h>
#include <time.h>
#include <algorithm>
#include <chrono>

#include "Window.h"
#include "Constants.h"
__device__ hiprandState* d_randStates;
// Kernel to initialize random states
__global__ void initRandStates(hiprandState* randStates, int width, int height, unsigned long long seed) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        hiprand_init(seed, idx, 0, &randStates[idx]);
    }
}

__global__ void initializeCellsKernel(Cell* cells, int width, int height, hiprandState* randStates) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        hiprandState localState = randStates[idx]; // Fetch pre-initialized state

        cells[idx].rotation = hiprand(&localState) % 4;// Randomly set energy
        cells[idx].energy = 0;// Randomly set energy
        cells[idx].activeGene = 0; // Initial age
        cells[idx].mutation = hiprand(&localState) % 1000;// Randomly set energy
        int isAlive = hiprand(&localState) % 20 == 0 ? 1 : 0; // Randomly set alive or dead      
        
        if (isAlive)
            cells[idx].energy = hiprand(&localState) % REP_ENERGY / 2;// Randomly set energy
        
        for (int g = 0; g < NUM_GENES; g++) 
            cells[idx].genes[g] = hiprand(&localState) % GENES; // Random genes
        

        randStates[idx] = localState; // Save state back
    }
}

__global__ void updateKernel(Cell* current, Cell* next, int width, int height, hiprandState* randStates) {
    
    int neighborOffsets[4][2] = {
       {0, -1}, // Up
       {0, 1},  // Down
       {-1, 0}, // Left
       {1, 0}   // Right
    };

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {

        int idx = y * width + x;
        hiprandState localState = randStates[idx]; // Fetch pre-initialized state

        //Copy Energy
        int energy = current[idx].energy;

        //Alive Cell
        if (energy > 0) {
            
            //Calculate Neibhors
            int neighborEnergy = 0;
            int neighborAlive = 0;
            int mutants = 0;
            int sharedEnergy = 0;
            int attackEnergy = 0;
            int damageEnergy = 0;

            for (int i = 0; i < 4; i++) {
                int neighborX = x + neighborOffsets[i][0];
                int neighborY = y + neighborOffsets[i][1];

                // Check if the neighbor is within bounds and alive
                if (neighborX >= 0 && neighborX < width && neighborY >= 0 && neighborY < height) {
                    if (current[neighborY * width + neighborX].energy > 0) {

                        neighborAlive++;
                        neighborEnergy += current[neighborY * width + neighborX].energy;
                        
                        //Neighbor Shared Energy
                        if (current[neighborY * width + neighborX].genes[current[neighborY * width + neighborX].activeGene] == 2)
                            sharedEnergy += current[neighborY * width + neighborX].energy / 5;
                        //Gain Energy From attack
                        if (current[neighborY * width + neighborX].energy)
                            attackEnergy -= current[neighborY * width + neighborX].energy;
                        else
                            attackEnergy += current[neighborY * width + neighborX].energy;
                        //Neighbor Attacked
                        if (current[neighborY * width + neighborX].genes[current[neighborY * width + neighborX].activeGene] == 4)
                            damageEnergy += current[neighborY * width + neighborX].energy;
                        if (current[neighborY * width + neighborX].genes[current[neighborY * width + neighborX].activeGene] == 12 &&
                            neighborOffsets[current[neighborY * width + neighborX].rotation][0] == -neighborOffsets[i][0] && 
                            neighborOffsets[current[neighborY * width + neighborX].rotation][1] == -neighborOffsets[i][1])
                            damageEnergy += current[neighborY * width + neighborX].energy;
                        if (current[neighborY * width + neighborX].genes[current[neighborY * width + neighborX].activeGene] == 13 &&
                            neighborOffsets[current[neighborY * width + neighborX].rotation][0] == -neighborOffsets[i][0] &&
                            neighborOffsets[current[neighborY * width + neighborX].rotation][1] == -neighborOffsets[i][1])
                            sharedEnergy += current[neighborY * width + neighborX].energy / 5;
                        //Mutant Detected
                        if (current[neighborY * width + neighborX].mutation != current[idx].mutation)
                            mutants++;
                    }
                }
            }
            //Remove Energy for reproduction
            if (neighborAlive > 0 && energy >= REP_ENERGY)
                energy = energy / neighborAlive;

            //Processed shared and damged energy
            //if(damageEnergy > energy)
            energy -= damageEnergy;

            energy += sharedEnergy;
            int newActiveGene = 0;

            //Update Gene
            next[idx].activeGene = (current[idx].activeGene + NUM_GENES + 1) % NUM_GENES;
            //Process Cells functions
            switch (current[idx].genes[current[idx].activeGene])
            {
            case 9:
                energy += REP_ENERGY / 100;

                break;
            case 1:
                energy -= 1;
                break;
            case 2:
                if (neighborAlive > 0)
                    energy /= neighborAlive + 1;
                energy -= 1;
                break;
            case 3:
                newActiveGene = (next[idx].activeGene + NUM_GENES + neighborAlive) % NUM_GENES;
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            case 4:
                energy += attackEnergy;
                break;
            case 5:
                newActiveGene = (next[idx].activeGene + NUM_GENES + mutants) % NUM_GENES;
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            case 6:
                newActiveGene = (next[idx].activeGene + NUM_GENES + 1) % NUM_GENES;
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            case 7:
                newActiveGene = (next[idx].activeGene + NUM_GENES + (NUM_GENES * neighborEnergy) / REP_ENERGY) % NUM_GENES;
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            case 8:
                newActiveGene = (next[idx].activeGene + NUM_GENES + (NUM_GENES * energy) / REP_ENERGY) % NUM_GENES;
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            case 10:
                newActiveGene = (next[idx].activeGene + NUM_GENES + (NUM_GENES * current[idx].rotation) / REP_ENERGY) % NUM_GENES;
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            case 11:
            {
                int neighborX = x + neighborOffsets[current[idx].rotation][0];
                int neighborY = y + neighborOffsets[current[idx].rotation][1];

                if (current[neighborY * width + neighborX].mutation == current[idx].mutation)
                    newActiveGene = (next[idx].activeGene + NUM_GENES + 1) % NUM_GENES;
                else
                    newActiveGene = (next[idx].activeGene + NUM_GENES + 2) % NUM_GENES;

                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            }
            case 12:
            {
                int neighborX = x + neighborOffsets[current[idx].rotation][0];
                int neighborY = y + neighborOffsets[current[idx].rotation][1];

                if (current[neighborY * width + neighborX].energy < energy)
                {
                    energy -= current[neighborY * width + neighborX].energy * 0;
                    newActiveGene = (next[idx].activeGene + NUM_GENES + 1) % NUM_GENES;
                }
                else
                {
                    energy += current[neighborY * width + neighborX].energy * 0;
                    newActiveGene = (next[idx].activeGene + NUM_GENES + 2) % NUM_GENES;
                }
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            }
            case 13:
            {
                int neighborX = x + neighborOffsets[current[idx].rotation][0];
                int neighborY = y + neighborOffsets[current[idx].rotation][1];

                energy = energy - energy / 5;
                newActiveGene = (next[idx].activeGene + NUM_GENES + 1) % NUM_GENES;
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            }
            case 14:
                next[idx].rotation = current[idx].genes[next[idx].activeGene] % 4;
                newActiveGene = (next[idx].activeGene + NUM_GENES + 1) % NUM_GENES;
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            case 15:
            {
                int neighborX = x + neighborOffsets[current[idx].rotation][0];
                int neighborY = y + neighborOffsets[current[idx].rotation][1];

                newActiveGene = (next[idx].activeGene + NUM_GENES + (NUM_GENES * current[neighborY * width + neighborX].energy) / REP_ENERGY) % NUM_GENES;
                next[idx].activeGene = current[idx].genes[newActiveGene];
                break;
            }
            }


            //random chance to die
            if (hiprand(&localState) % 10000 == 0)
                energy = 0;

            // remove energy for stayin alive
            energy -= REP_ENERGY / 500;

            //Check if the energy is in the proper range
            if (energy <= 0 || energy>=2* REP_ENERGY)
                energy = 0;

        }
        //Empty Cell
        else {
            int parents[4];
            int parentAlive = 0;
            int parentEnergy = 0;

            //Calculate Parents
            for (int i = 0; i < 4; i++) {
                int neighborX = x + neighborOffsets[i][0];
                int neighborY = y + neighborOffsets[i][1];

                // Check if the neighbor is within bounds and alive
                if (neighborX >= 0 && neighborX < width && neighborY >= 0 && neighborY < height) {
                    if (current[neighborY * width + neighborX].energy > REP_ENERGY) {
                        parents[parentAlive] = i;
                        parentAlive++;
                        parentEnergy += current[neighborY * width + neighborX].energy;
                    }
                }
            }
            
            //Process Reproduction
            if (parentAlive > 0)
            {

                //Copy energy of parents
                energy = parentEnergy / parentAlive / 2 - 1;
                energy = energy > 0 ? energy : 0;
                //Copy Genes
                int j = parents[hiprand(&localState) % parentAlive];
                int neighborX = x + neighborOffsets[j][0];
                int neighborY = y + neighborOffsets[j][1];

                next[idx].rotation = current[neighborY * width + neighborX].rotation;
                current[idx].rotation = current[neighborY * width + neighborX].rotation;
                next[idx].mutation = current[neighborY * width + neighborX].mutation;
                current[idx].mutation = current[neighborY * width + neighborX].mutation;

                for (int i = 0; i < NUM_GENES; ++i)
                {
                    next[idx].genes[i] = current[neighborY * width + neighborX].genes[i];
                    current[idx].genes[i] = current[neighborY * width + neighborX].genes[i];
                }
                //Mutation
                if (hiprand(&localState) % 40 == 0)
                {
                    next[idx].mutation = hiprand(&localState) % 100000;
                    current[idx].mutation = next[idx].mutation;
                    int gn = hiprand(&localState) % NUM_GENES;
                    int gv = hiprand(&localState) % GENES;
                    next[idx].genes[gn] = gv;
                    current[idx].genes[gn] = gv;
                }

                //Set ActiveGene to 0
                next[idx].activeGene = 0;
            }
        }
        next[idx].energy = energy;

        randStates[idx] = localState; // Save state back
    }
}


int main() {
    if (true)
    {
        // Prompt the user to enter the width
        std::cout << "Enter the width: ";
        std::cin >> WIDTH;

        // Prompt the user to enter the height
        std::cout << "Enter the height: ";
        std::cin >> HEIGHT;

        // Optionally, you can print the values to confirm
        std::cout << "Width set to: " << WIDTH << std::endl;
        std::cout << "Height set to: " << HEIGHT << std::endl;
    }
    else
    {
        WIDTH = 200;
        HEIGHT = 200;
    }
    int cellSize = 1;
    Window window(WIDTH * cellSize, HEIGHT * cellSize, "Game of Life");

    Cell* current = (Cell*)malloc(WIDTH * HEIGHT * sizeof(Cell));

    srand(time(NULL));

    Cell* dev_current = 0;
    Cell* dev_next = 0;
    hipError_t cudaStatus;

    // Allocate GPU buffers for Cell arrays
    cudaStatus = hipMalloc((void**)&dev_current, WIDTH * HEIGHT * sizeof(Cell));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }

    cudaStatus = hipMalloc((void**)&dev_next, WIDTH * HEIGHT * sizeof(Cell));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return 1;
    }
  

    // Initialize cells on the GPU
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    hipMalloc(&d_randStates, WIDTH * HEIGHT * sizeof(hiprandState));
    initRandStates << <numBlocks, threadsPerBlock >> > (d_randStates, WIDTH, HEIGHT, time(NULL));
    hipDeviceSynchronize();

    // Use a seed for random number generation
    initializeCellsKernel << <numBlocks, threadsPerBlock >> > (dev_current, WIDTH, HEIGHT, d_randStates);
    hipDeviceSynchronize();
    hipMemcpy(dev_next, dev_current, WIDTH * HEIGHT * sizeof(Cell), hipMemcpyDeviceToDevice);

    // Mouse state variables
    double mouseX, mouseY;
    int mouseButtonState;

    int a = 200;
    bool isUpdateFrame = true;
    auto lastTime = std::chrono::high_resolution_clock::now(); // Start time
    int frameCount = 0; // Frame counter

    int generationCounter=0;
    // Main loop
    while (!window.shouldClose()) {

        // Check mouse events
        {
            glfwGetCursorPos(window.window, &mouseX, &mouseY); // Get mouse position
            mouseButtonState = glfwGetMouseButton(window.window, GLFW_MOUSE_BUTTON_LEFT); // Left mouse button state

            if (mouseButtonState == GLFW_PRESS && isUpdateFrame) {
                // Convert mouse position to grid coordinates
                int gridX = (int)(mouseX / window.cellWidth);
                int gridY = HEIGHT - (int)(mouseY / window.cellHeight); // Flip Y axis since OpenGL has the origin at the bottom left
                int radius = 50;
                // Ensure the coordinates are within grid bounds
                if (gridX >= 0 && gridX < WIDTH && gridY >= 0 && gridY < HEIGHT) {
                    // Set cells within the radius to alive
                    for (int dx = -radius; dx <= radius; dx++) {
                        for (int dy = -radius; dy <= radius; dy++) {
                            // Calculate the distance from the center
                            if (dx * dx + dy * dy <= radius * radius) {
                                int newX = gridX + dx;
                                int newY = gridY + dy;

                                // Ensure the new coordinates are within grid bounds
                                if (newX >= 0 && newX < WIDTH && newY >= 0 && newY < HEIGHT) {
                                    // Set the cell to alive at the calculated position
                                    current[newY * WIDTH + newX].energy = REP_ENERGY / 2;
                                    for (int i = 0; i < NUM_GENES; ++i)
                                    {
                                        //TO DO make the genes random value from 0 to GENES
                                        current[newY * WIDTH + newX].genes[i] = rand() % (GENES); // Random value from 0 to GENES
                                    }
                                    printf("Cell alive at (%d, %d)\n", newX, newY);
                                }
                            }
                        }
                    }
                    hipMemcpy(dev_current, current, WIDTH * HEIGHT * sizeof(Cell), hipMemcpyHostToDevice);
                }
            }
        }
        // Check keyboard events
        {
            if (glfwGetKey(window.window, GLFW_KEY_O) == GLFW_PRESS) {
                a = 1; // Increase a by 1 when 'A' is pressed
                printf("Variable set to: %d\n", a);
            }
            if (glfwGetKey(window.window, GLFW_KEY_I) == GLFW_PRESS) {
                std::cout << "Enter generation per frame ( currently" << a << ") :";
                std::cin >> a;
                if (a <0)
                {
                    a = -a;
                    isUpdateFrame = !isUpdateFrame;
                    window.setGeneration(generationCounter);
                    window.showSideBar();
                }
                else
                {
                    window.hideSideBar();
                }
                printf("Variable set to: %d\n", a);
            }
            if (glfwGetKey(window.window, GLFW_KEY_A) == GLFW_PRESS) {
                a++; // Increase a by 1 when 'A' is pressed
                printf("Variable a increased to: %d\n", a);
            }
            if (glfwGetKey(window.window, GLFW_KEY_Q) == GLFW_PRESS) {
                a--; // Decrease a by 1 when 'Q' is pressed
                if (a < 0)
                    a = 0;
                printf("Variable a decreased to: %d\n", a);
            }

            if (glfwGetKey(window.window, GLFW_KEY_R) == GLFW_PRESS ) {
                initializeCellsKernel << <numBlocks, threadsPerBlock >> > (dev_current, WIDTH, HEIGHT, d_randStates);
                hipDeviceSynchronize();
                printf("Reset\n");
            }
        }

        // Run the simulation for a number of generations
        for (int generation = 0; generation < a; generation++) {
            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x,
                (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

            unsigned long long seed = time(NULL)  + generation; // Change seed for each generation
            updateKernel << <numBlocks, threadsPerBlock >> > (dev_current, dev_next, WIDTH, HEIGHT, d_randStates);
           
            hipDeviceSynchronize();

            // Swap the buffers
            Cell* temp = dev_current;
            dev_current = dev_next;
            dev_next = temp;
        }

        generationCounter += a;

        if (isUpdateFrame)
        {
            // Copy the current generation back to the host
            hipMemcpy(current, dev_current, WIDTH * HEIGHT * sizeof(Cell), hipMemcpyDeviceToHost);
            window.renderGrid(current);
            window.swapBuffers();

        }
        else
        {
            window.setGeneration(generationCounter);
            window.renderSidebar();
            window.swapBuffers();

        }
            window.pollEvents();

        // Frame rate calculation
        frameCount+= a;
        auto currentTime = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = currentTime - lastTime;

        // Calculate time per frame
        std::chrono::duration<double> frameTime = currentTime - lastTime; // Time taken for the current frame

        if (elapsed.count() >= 1.0) { // If one second has passed
            std::cout << "FPS: " << frameCount / elapsed.count() << std::endl; // Display the frame rate
            std::cout << "Time per frame: " << frameTime.count() << " seconds" << std::endl; // Display time per frame
            frameCount = 0; // Reset the frame count
            lastTime = currentTime; // Update the last time
        }
    }

    // Clean up
    hipFree(dev_current);
    hipFree(dev_next);
    free(current);

    return 0;
}
